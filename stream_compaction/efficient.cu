#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int* data, int offset) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);

            // d = 1
            // 4
            int multiple = 1 << (offset + 1);

            int newIndex = index * multiple;

            if (newIndex > n - 1) {
                return;
            }

            // bit shift equivalent to power of 2
            int base = multiple >> 1;

            // going left to right now
            data[newIndex + multiple - 1] += data[newIndex + base - 1];            
        }

        __global__ void kernDownSweep(int n, int* data, int offset) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);

            int multiple = 1 << (offset + 1);

            int newIndex = index * multiple;

            if (newIndex > n - 1) {
                return;
            }

            // bit shift equivalent to power of 2
            int base = multiple >> 1;

            int leftChild = data[newIndex + base - 1];
            // setting left as right child's val
            data[newIndex + base - 1] = data[newIndex + multiple - 1];
            // adding left child to right
            data[newIndex + multiple - 1] += leftChild;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO

            int padding = 1 << ilog2ceil(n);

            int blockSize = 128;
            dim3 fullBlocksPerGrid((padding + blockSize - 1) / blockSize);

            // read & write buffer b/c no overlap now
            int* dev_data;

            // CUDA memory management and error checking.
            hipMalloc((void**)&dev_data, padding * sizeof(int));
            checkCUDAError("hipMalloc data failed!");

            hipMemset(dev_data, 0, padding * sizeof(int));

            // copying idata into buffer
            hipMemcpy(dev_data + padding - n, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            for (int d = 0; d <= ilog2ceil(padding) - 1; d++) {
                // typical CUDA kernel invocation.
                kernUpSweep <<< fullBlocksPerGrid, blockSize >>> (padding, dev_data, d);
                checkCUDAError("UpSweep failed!");

                // synchronize
                hipDeviceSynchronize();
            }

            // put zero in root (last elem of dev_data)
            int zerotest[1] = { 0 }; // check if this or int?
            hipMemcpy(dev_data + padding - 1, zerotest, sizeof(int), hipMemcpyHostToDevice);

            for (int d = ilog2ceil(padding) - 1; d >= 0; d--) {
                kernDownSweep <<< fullBlocksPerGrid, blockSize>>> (padding, dev_data, d);
                checkCUDAError("DownSweep failed!");

                hipDeviceSynchronize();
            }

            timer().endGpuTimer();
        
            hipMemcpy(odata, dev_data + padding - n, sizeof(int) * n, hipMemcpyDeviceToHost);

            hipFree(dev_data);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {

            int padding = 1 << ilog2ceil(n);

            int blockSize = 128;
            dim3 fullBlocksPerGrid((padding + blockSize - 1) / blockSize);

            int* dev_indices;
            int* dev_bools;
            int* dev_out;
            int* dev_in;

            // CUDA memory management and error checking.
            hipMalloc((void**)&dev_indices, padding * sizeof(int));
            checkCUDAError("hipMalloc indices failed!");
            
            hipMalloc((void**)&dev_bools, padding * sizeof(int));
            checkCUDAError("hipMalloc bools failed!");

            hipMalloc((void**)&dev_in, padding * sizeof(int));
            checkCUDAError("hipMalloc in failed!");

            hipMalloc((void**)&dev_out, padding * sizeof(int));
            checkCUDAError("hipMalloc out failed!");

            hipMemset(dev_in, 0, padding * sizeof(int));
            hipMemcpy(dev_in + padding - n, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            timer().startGpuTimer();
            // TODO
            Common::kernMapToBoolean << < fullBlocksPerGrid, blockSize >> > (padding, dev_bools, dev_in);
            checkCUDAError("MapToBool failed!");

            // need to do scan on temp array
            hipMemcpy(dev_indices, dev_bools, sizeof(int) * padding, hipMemcpyDeviceToDevice);

            for (int d = 0; d <= ilog2ceil(padding) - 1; d++) {
                // typical CUDA kernel invocation.
                kernUpSweep << < fullBlocksPerGrid, blockSize >> > (padding, dev_indices, d);
                checkCUDAError("UpSweep failed!");

                // synchronize
                hipDeviceSynchronize();
            }

            // put zero in root (last elem of dev_data)
            int zerotest[1] = { 0 }; // check if this or int?
            hipMemcpy(dev_indices + padding - 1, zerotest, sizeof(int), hipMemcpyHostToDevice);

            for (int d = ilog2ceil(padding) - 1; d >= 0; d--) {
                kernDownSweep << < fullBlocksPerGrid, blockSize >> > (padding, dev_indices, d);
                checkCUDAError("DownSweep failed!");

                hipDeviceSynchronize();
            }

            Common::kernScatter << < fullBlocksPerGrid, blockSize >> > (n, dev_out,
                dev_in, dev_bools, dev_indices);
            
            timer().endGpuTimer();
            // size is in last elem of indices
            int size[1];
            int sizeTest[1];

            hipMemcpy(size, dev_indices + padding - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(sizeTest, dev_bools + padding - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(odata, dev_out + padding - n, sizeof(int) * n, hipMemcpyDeviceToHost);

            hipFree(dev_in);
            hipFree(dev_bools);
            hipFree(dev_indices);
            hipFree(dev_out);

            int returnval = size[0] + sizeTest[0];

            return returnval;
        }
    }
}
