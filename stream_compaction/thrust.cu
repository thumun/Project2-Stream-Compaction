#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            thrust::host_vector<int> h_in(n, &idata);
            thrust::host_vector<int> h_out(n);
            thrust::device_vector<int> dev_thrust_input = h_in;
            thrust::device_vector<int> dev_thrust_output = h_out;

            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
            thrust::exclusive_scan(dev_thrust_input.begin(), dev_thrust_input.end(), dev_thrust_output.begin());
            timer().endGpuTimer();

            thrust::copy(dev_thrust_output.begin(), dev_thrust_output.end(), odata);
        }
    }
}
